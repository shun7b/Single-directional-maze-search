#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA

#define BS 1024
#define ROAD 0
#define WIDTH 8 
#define HEIGHT  8 
#define MAPSIZE (WIDTH*HEIGHT) 
#define WALL (MAPSIZE*16 -1)
//#define N  524288/* 配列の長さ、2の30乗 
 /* GPUカーネル関数の定義*/
 __global__ void bidirectional_search(int *Dend,int *j,int *DA,int *g,int *start,int *goal,int *end_sg)
 {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i>=(((MAPSIZE)))){
    return;
	}
	if(*end_sg>0){	
		return;
	}
	if((DA[i]!=WALL)&&((i)!=(*goal))){
if(((g[i])==0)&&(g[i+1]|g[i-1]|g[i+WIDTH]|g[i-WIDTH])){
					Dend[i]=(*j);
					if(i==(*start)){
						*end_sg=1;
					}
		}
			g[i]=g[i+1]|g[i-1]|g[i+WIDTH]|g[i-WIDTH];
	
	}
return;
}
int main(void)
{
 int *i;
// int j;
 int *start_gpu,*goal_gpu;	
 int *start,*goal;	
 int *start_flag,*goal_flag;
 int *Dgoal_flag;
 int *ROUTE;
 int *A; /* ホストメモリ用のポインタ*/
 long start_time,end_time,pre_time_start,pre_time_end;
 int *ii;
 int *end,*gend;
 int *wall_end;
 int *Dwall_end;
 int j=0;
 wall_end = (int *)malloc((MAPSIZE) *sizeof(int));
     // 各行ごとに列数分のメモリを確保
  ROUTE=(int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
  goal_flag = (int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
  start = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  goal = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  i = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  start_flag = (int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
  end = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  *end=0;
  *start=WIDTH+1;
  *goal=(MAPSIZE)-WIDTH-2;
  start_flag[*start]=1;
  goal_flag[*goal]=1;
  	  for(int in=0;in<(MAPSIZE);in++){
		if((in/WIDTH)==0||(in/WIDTH)==HEIGHT-1){
				ROUTE[in]=WALL;
		}else if((in%WIDTH)==0||(in%WIDTH)==WIDTH-1){
				ROUTE[in]=WALL;
		}else if((in/WIDTH)%2==1){
				ROUTE[in]=ROAD;
		}else if((in/WIDTH)%2==0){
			if((in%WIDTH)%2==0){
				ROUTE[in]=ROAD;
			}else{
				ROUTE[in]=WALL;
			}
		}
		wall_end[in]=WALL;
	}
		wall_end[*goal]=0;
 pre_time_start=clock();
 hipMalloc((int**)&Dwall_end, sizeof(int)*MAPSIZE); 
	//printf("s%04d,g%d\n",*start,*goal);
 hipMalloc((int**)&ii, sizeof(int)); 
 hipMalloc((int**)&A, sizeof(int)*(MAPSIZE)); 
 hipMalloc((int**)&Dgoal_flag, sizeof(int)*(MAPSIZE)); 
 hipMalloc((int**)&start_gpu, sizeof(int)); 
 hipMalloc((int**)&goal_gpu, sizeof(int)); 
 hipMalloc((int**)&gend, sizeof(int)); 
 hipMemcpy( start_gpu,start, sizeof(int), hipMemcpyDefault);
 hipMemcpy( goal_gpu,goal, sizeof(int), hipMemcpyDefault);
 pre_time_end=clock();
	//printf("pre_time=%ld\n",pre_time_end-pre_time_start);
	for(*i=0;*i<(MAPSIZE);*i=(*i+1)){	
		//printf("%4d,",ROUTE[*i]);
		if((*i%WIDTH)==(WIDTH-1)){
			//printf("\n");
		}
	}
	start_time=clock();
	for(*i=1;(*i)<MAPSIZE;*i=(*i+1)){
 		hipMemcpy( Dwall_end,wall_end, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( ii,i, sizeof(int), hipMemcpyDefault);
 		hipMemcpy( A,ROUTE, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( Dgoal_flag,goal_flag, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( gend,end ,sizeof(int), hipMemcpyDefault);
		bidirectional_search<<<((MAPSIZE)+BS-1)/BS, BS,2>>>(Dwall_end  ,ii,A,Dgoal_flag,start_gpu,goal_gpu,gend);
		hipDeviceSynchronize();
 		hipMemcpy( wall_end,Dwall_end, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( ROUTE,A, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( goal_flag,Dgoal_flag, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( end,gend ,sizeof(int), hipMemcpyDefault);
		//printf("end------------\n");
		if(*end==1){
				j++;
				if(j>1){
				break;
			}
}
	for(int in=0;in<(MAPSIZE);in=(in+1)){	
				if((in%WIDTH)==(WIDTH-1)){
					//printf("\n");
				}
		if(*end!=1){
		}
	}
}
		wall_end[*goal]=0;
		//printf("end------------%d\n",*i);
	end_time=clock();
	for(*i=0;*i<(MAPSIZE);*i=(*i+1)){	
		//printf("%4d,",wall_end[*i]);
		ROUTE[*i]=wall_end[*i];
		if(((*i)%WIDTH)==(WIDTH-1)){
			//printf("\n");
		}
	}
	hipFree(A);
	hipFree(Dwall_end);
	hipFree(start_gpu);
	hipFree(ii);
	hipFree(goal_gpu);
	hipFree(gend);
	hipFree(Dwall_end);
	free(i);
	free(end);
	free(wall_end);
	//printf("s%04d,g%d\n",*start,*goal);
	//printf("exe_time=%ld\n,",(end_time-start_time));
	if(0){
		int nowplot,nextplot;
		int count;
		int *result;
		  result= (int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
					nowplot=(*start);
					nextplot=(*start);
						
					count=0;
					while(nowplot!=(*goal)){
						//printf("%d ", nowplot);
						result[count]=nowplot;
						if(ROUTE[nowplot-1]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot-1]){
								nextplot=nowplot-1;
							}
						}
						if(ROUTE[nowplot+1]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot+1]){
								nextplot=nowplot+1;
							}
						}
						if(ROUTE[nowplot+WIDTH]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot+WIDTH]){
								nextplot=nowplot+WIDTH;
							}
						}
						if(ROUTE[nowplot-WIDTH]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot-WIDTH]){
								nextplot=nowplot-WIDTH;
							}
						}
						nowplot=nextplot;        
						count++;
					}
					result[count]=(*goal);
					count=0;
					while(result[count]!=(*goal)){
						//printf(" %d ",result[count]);
						count++;
					}
	}	
	free(ROUTE);
	free(start);
	free(goal);
	printf("exe_time=%ld\n,",(end_time-start_time));
	return 0;
 }
